#include "hip/hip_runtime.h"

#include "CudaOCTProcess.cuh"

#ifdef CUDA_ENABLED

#include <cuda/math_functions.h>
#include "Common/CudaErrorCheck.cuh"


// Constant Memory for Calibration Data
#define _64KB_ 64 * 1024
__device__ __constant__ unsigned char cudaConstMem[_64KB_];


// GPU kernels for sub-functions
__device__ void complexMul(hipComplex* pDst, hipComplex* pSrc1, hipComplex* pSrc2)
{
	(*pDst).x = (*pSrc1).x * (*pSrc2).x - (*pSrc1).y * (*pSrc2).y;
	(*pDst).y = (*pSrc1).x * (*pSrc2).y + (*pSrc1).y * (*pSrc2).x;
}

__global__ void preprocessing(float* fringeF32, ushort* fringeU16, int width, int widthFFT)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x < width)
	{
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		int offset0 = x + y * width;
		int offset1 = x + y * widthFFT; // Automatical zero-padding

		float* bg = (float*)cudaConstMem; // OCT interferogram background (from constant memory)
		float* win = (float*)cudaConstMem + width; // Hanning window (from constant memory)

		// single-precision conversion & zero-padding
		// Background subtraction & hann windowing
		fringeF32[offset1] = win[x] * ((float)fringeU16[offset0] - bg[x]);		
	}
}

__global__ void demodulation(hipComplex* pDemodul, hipComplex* pModul, int width)
{
#ifdef FREQ_SHIFTING
	int x0 = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int x1 = ((width - x0) + width / 4) % width;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int offset0 = x0 + y * width;
	int offset1 = x1 + y * width;

	pDemodul[offset1].x = (pModul[offset0].x) / width;
	pDemodul[offset1].y = (-pModul[offset0].y) / width;
#else
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int offset = x + y * width;

	if (x < width / 2)
	{
		pDemodul[offset].x = pModul[offset].x / width;
		pDemodul[offset].y = pModul[offset].y / width;
	}
	else
	{
		pDemodul[offset].x = 0;
		pDemodul[offset].y = 0;
	}
#endif
}

__global__ void calibration(hipComplex* pCalib, hipComplex* pSignal, int width, int widthFFT)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x < width)
	{
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		float* calib_idx = (float*)cudaConstMem + 4 * width; // k-linearization index (from constant memory)
		float* calib_weight = (float*)cudaConstMem + 5 * width; // k-linearization weight (from constant memory)
		hipComplex* disp_comp = (hipComplex*)cudaConstMem + 3 * width; // dispersion compensation (from constant memory)

		int offset0 = x + y * widthFFT / 2;
		int offset1 = (int)calib_idx[x] + y * widthFFT;

		// k-linearization
		hipComplex temp;
		temp.x = calib_weight[x] * pSignal[offset1].x + (1 - calib_weight[x]) * pSignal[offset1 + 1].x;
		temp.y = calib_weight[x] * pSignal[offset1].y + (1 - calib_weight[x]) * pSignal[offset1 + 1].y;

		// dispersion compensation
		complexMul(&pCalib[offset0], &temp, &disp_comp[x]);
	}
}

__global__ void logScaling(float* pScaled, hipComplex* pComplex, int width, bool is_linear)
{
	int x0 = threadIdx.x + blockIdx.x * blockDim.x;
#ifdef FREQ_SHIFTING
	int x1 = (x0 + width / 2) % width;
#else
	int x1 = x0;
#endif
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int offset0 = x0 + y * blockDim.x * gridDim.x;
	int offset1 = x1 + y * blockDim.x * gridDim.x;

	// log Scaling
	if (!is_linear)
		pScaled[offset1] = 10 * __log10f(pComplex[offset0].x * pComplex[offset0].x + pComplex[offset0].y * pComplex[offset0].y);
	else
		pScaled[offset1] = (pComplex[offset0].x * pComplex[offset0].x + pComplex[offset0].y * pComplex[offset0].y);
}



// Class member function definition
CudaOCTProcess:: CudaOCTProcess(int _nScans, int _nAlines) :
	OCTProcess(_nScans, _nAlines),
	nScans(_nScans), n2Scans(_nScans / 2),
	nScansFFT((int)(1 << (int)ceil(log2(_nScans)))),
	n2ScansFFT((int)(1 << (int)ceil(log2(_nScans / 2)))),
	nAlines(_nAlines)
{
}

CudaOCTProcess::~CudaOCTProcess()
{
	// Free Memories
	freeMemory();
}



void CudaOCTProcess::operator()(float* img, uint16_t* fringe)
{
    int transfer_nAlines = nAlines / N_CUDA_STREAMS / N_CUDA_PARTITIONS;
    for (int i = 0; i < N_CUDA_PARTITIONS; i++)
	{
		// Transfer to Device ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			CUDA_CHECK_ERROR(hipMemcpyAsync(deviceRawFringeU16 + nScans * j * transfer_nAlines, (ushort*)fringe + nScans * (i + N_CUDA_PARTITIONS * j) * transfer_nAlines,
				sizeof(ushort) * nScans * transfer_nAlines, hipMemcpyHostToDevice, stream[j]));

		// Preprocessing /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        for (int j = 0; j < N_CUDA_STREAMS; j++)
			preprocessing << < blocksPerGrid[0], threadsPerBlock, 0, stream[j] >> >
				(deviceRawFringeF32 + nScansFFT * j * transfer_nAlines, deviceRawFringeU16 + nScans * j * transfer_nAlines, nScans, nScansFFT);

		// Real-To-Complex FFT ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        for (int j = 0; j < N_CUDA_STREAMS; j++)
			CUFFT_CHECK_ERROR(hipfftExecR2C(cufftPlan_R2C[j],
				deviceRawFringeF32 + nScansFFT * j * transfer_nAlines, deviceBScan + nScansFFT * j * transfer_nAlines));

		// Demodulation ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        for (int j = 0; j < N_CUDA_STREAMS; j++)
			demodulation << < blocksPerGrid[1], threadsPerBlock, 0, stream[j] >> >
				(deviceDemodulBScan + nScansFFT * j * transfer_nAlines, deviceBScan + nScansFFT * j * transfer_nAlines, nScansFFT);

		// Complex-To-Complex IFFT ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        for (int j = 0; j < N_CUDA_STREAMS; j++)
			CUFFT_CHECK_ERROR(hipfftExecC2C(cufftPlan_C2C_Inv[j],
				deviceDemodulBScan + nScansFFT * j * transfer_nAlines, deviceDemodulSignal + nScansFFT * j * transfer_nAlines, HIPFFT_BACKWARD));

		// k-linearization & Dispersion Compensation //////////////////////////////////////////////////////////////////////////////////////////////////////
        for (int j = 0; j < N_CUDA_STREAMS; j++)
			calibration << < blocksPerGrid[2], threadsPerBlock, 0, stream[j] >> >
				(deviceCalibSignal + n2ScansFFT * j * transfer_nAlines, deviceDemodulSignal + nScansFFT * j * transfer_nAlines, n2Scans, nScansFFT);

		// Complex-To-Complex FFT ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        for (int j = 0; j < N_CUDA_STREAMS; j++)
			CUFFT_CHECK_ERROR(hipfftExecC2C(cufftPlan_C2C_Fwd[j],
				deviceCalibSignal + n2ScansFFT * j * transfer_nAlines, deviceCalibBScan + n2ScansFFT * j * transfer_nAlines, HIPFFT_FORWARD));

		// Scaling ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////		
        for (int j = 0; j < N_CUDA_STREAMS; j++)
			logScaling << < blocksPerGrid[1], threadsPerBlock, 0, stream[j] >> >
				(deviceScaledBScan + n2ScansFFT * j * transfer_nAlines, deviceCalibBScan + n2ScansFFT * j * transfer_nAlines, n2ScansFFT);

		// Transfer to Host //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        for (int j = 0; j < N_CUDA_STREAMS; j++)
            CUDA_CHECK_ERROR(hipMemcpyAsync(img + n2ScansFFT * (i + N_CUDA_PARTITIONS * j) * transfer_nAlines, deviceScaledBScan + n2ScansFFT * j * transfer_nAlines,
				sizeof(float) * n2ScansFFT * transfer_nAlines, hipMemcpyDeviceToHost, stream[j]));
	}
    for (int i = 0; i < N_CUDA_STREAMS; i++)
		CUDA_CHECK_ERROR(hipStreamSynchronize(stream[i]));
}

void CudaOCTProcess::operator()(float* img, uint16_t* fringe, const char* linear)
{
	int transfer_nAlines = nAlines / N_CUDA_STREAMS / N_CUDA_PARTITIONS;
	for (int i = 0; i < N_CUDA_PARTITIONS; i++)
	{
		// Transfer to Device ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			CUDA_CHECK_ERROR(hipMemcpyAsync(deviceRawFringeU16 + nScans * j * transfer_nAlines, (ushort*)fringe + nScans * (i + N_CUDA_PARTITIONS * j) * transfer_nAlines,
				sizeof(ushort) * nScans * transfer_nAlines, hipMemcpyHostToDevice, stream[j]));

		// Preprocessing /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			preprocessing << < blocksPerGrid[0], threadsPerBlock, 0, stream[j] >> >
				(deviceRawFringeF32 + nScansFFT * j * transfer_nAlines, deviceRawFringeU16 + nScans * j * transfer_nAlines, nScans, nScansFFT);

		// Real-To-Complex FFT ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			CUFFT_CHECK_ERROR(hipfftExecR2C(cufftPlan_R2C[j],
				deviceRawFringeF32 + nScansFFT * j * transfer_nAlines, deviceBScan + nScansFFT * j * transfer_nAlines));

		// Demodulation ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			demodulation << < blocksPerGrid[1], threadsPerBlock, 0, stream[j] >> >
				(deviceDemodulBScan + nScansFFT * j * transfer_nAlines, deviceBScan + nScansFFT * j * transfer_nAlines, nScansFFT);

		// Complex-To-Complex IFFT ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			CUFFT_CHECK_ERROR(hipfftExecC2C(cufftPlan_C2C_Inv[j],
				deviceDemodulBScan + nScansFFT * j * transfer_nAlines, deviceDemodulSignal + nScansFFT * j * transfer_nAlines, HIPFFT_BACKWARD));

		// k-linearization & Dispersion Compensation //////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			calibration << < blocksPerGrid[2], threadsPerBlock, 0, stream[j] >> >
				(deviceCalibSignal + n2ScansFFT * j * transfer_nAlines, deviceDemodulSignal + nScansFFT * j * transfer_nAlines, n2Scans, nScansFFT);

		// Complex-To-Complex FFT ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			CUFFT_CHECK_ERROR(hipfftExecC2C(cufftPlan_C2C_Fwd[j],
				deviceCalibSignal + n2ScansFFT * j * transfer_nAlines, deviceCalibBScan + n2ScansFFT * j * transfer_nAlines, HIPFFT_FORWARD));

		// Scaling ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////		
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			logScaling << < blocksPerGrid[1], threadsPerBlock, 0, stream[j] >> >
				(deviceScaledBScan + n2ScansFFT * j * transfer_nAlines, deviceCalibBScan + n2ScansFFT * j * transfer_nAlines, n2ScansFFT, true);

		// Transfer to Host //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for (int j = 0; j < N_CUDA_STREAMS; j++)
			CUDA_CHECK_ERROR(hipMemcpyAsync(img + n2ScansFFT * (i + N_CUDA_PARTITIONS * j) * transfer_nAlines, deviceScaledBScan + n2ScansFFT * j * transfer_nAlines,
				sizeof(float) * n2ScansFFT * transfer_nAlines, hipMemcpyDeviceToHost, stream[j]));
	}
	for (int i = 0; i < N_CUDA_STREAMS; i++)
		CUDA_CHECK_ERROR(hipStreamSynchronize(stream[i]));
	
	(void)linear;
}


void CudaOCTProcess::initialize()
{
	transferCalibData();
	allocateMemory();
	setGridBlockDimension();
	setCudaStream();
	setCufftPlan();
}


void CudaOCTProcess::transferCalibData()
{
	// Transfer OCT Calibration Data to Device as Constant Memory
	int tempOffset = 0;
	unsigned char* temp = (unsigned char*)malloc(sizeof(float) * 4 * nScans);
	memcpy(temp + tempOffset, this->getBg(), sizeof(float) * nScans);  tempOffset += sizeof(float) * nScans;
	memcpy(temp + tempOffset, this->getWin(), sizeof(float) * nScans);  tempOffset += sizeof(float) * nScans;
	memcpy(temp + tempOffset, this->getCalibIndex(), sizeof(float) * n2Scans); tempOffset += sizeof(float) * n2Scans;
	memcpy(temp + tempOffset, this->getCalibWeight(), sizeof(float) * n2Scans); tempOffset += sizeof(float) * n2Scans;
	memcpy(temp + tempOffset, this->getDispComp(), sizeof(hipComplex) * n2Scans); tempOffset += sizeof(hipComplex) * n2Scans;

	CUDA_CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstMem), temp, tempOffset)); free(temp);
}


void CudaOCTProcess::allocateMemory()
{
	// Set Host & Device Memories
    CUDA_CHECK_ERROR(hipMalloc((void**)&deviceRawFringeU16, sizeof(ushort2) * nScans * nAlines / N_CUDA_PARTITIONS));
    CUDA_CHECK_ERROR(hipMalloc((void**)&deviceRawFringeF32, sizeof(float) * nScansFFT * nAlines / N_CUDA_PARTITIONS));
    CUDA_CHECK_ERROR(hipMemset(deviceRawFringeF32, 0, sizeof(float) * nScansFFT * nAlines / N_CUDA_PARTITIONS));

    CUDA_CHECK_ERROR(hipMalloc((void**)&deviceBScan, sizeof(hipComplex) * nScansFFT * nAlines / N_CUDA_PARTITIONS));
    CUDA_CHECK_ERROR(hipMalloc((void**)&deviceDemodulBScan, sizeof(hipComplex) * nScansFFT * nAlines / N_CUDA_PARTITIONS));
    CUDA_CHECK_ERROR(hipMemset(deviceBScan, 0, sizeof(hipComplex) * nScansFFT * nAlines / N_CUDA_PARTITIONS));
    CUDA_CHECK_ERROR(hipMemset(deviceDemodulBScan, 0, sizeof(hipComplex) * nScansFFT * nAlines / N_CUDA_PARTITIONS));

    CUDA_CHECK_ERROR(hipMalloc((void**)&deviceDemodulSignal, sizeof(hipComplex) * nScansFFT * nAlines / N_CUDA_PARTITIONS));
    CUDA_CHECK_ERROR(hipMalloc((void**)&deviceCalibSignal, sizeof(hipComplex) * n2ScansFFT * nAlines / N_CUDA_PARTITIONS));
    CUDA_CHECK_ERROR(hipMemset(deviceCalibSignal, 0, sizeof(hipComplex) * n2ScansFFT * nAlines / N_CUDA_PARTITIONS));

    CUDA_CHECK_ERROR(hipMalloc((void**)&deviceCalibBScan, sizeof(hipComplex) * n2ScansFFT * nAlines / N_CUDA_PARTITIONS));
    CUDA_CHECK_ERROR(hipMalloc((void**)&deviceScaledBScan, sizeof(float) * n2ScansFFT * nAlines / N_CUDA_PARTITIONS));
}

void CudaOCTProcess::setGridBlockDimension()
{
	// Grid and Block Dimension 
    blocksPerGrid[0] = dim3((nScans + N_CUDA_THREADS - 1) / N_CUDA_THREADS, nAlines / N_CUDA_PARTITIONS / N_CUDA_STREAMS / N_CUDA_THREADS);
    blocksPerGrid[1] = dim3((n2ScansFFT + N_CUDA_THREADS - 1) / N_CUDA_THREADS, nAlines / N_CUDA_PARTITIONS / N_CUDA_STREAMS / N_CUDA_THREADS);
    blocksPerGrid[2] = dim3((n2Scans + N_CUDA_THREADS - 1) / N_CUDA_THREADS, nAlines / N_CUDA_PARTITIONS / N_CUDA_STREAMS / N_CUDA_THREADS);
    threadsPerBlock = dim3(N_CUDA_THREADS, N_CUDA_THREADS);
}

void CudaOCTProcess::setCudaStream()
{
	// CUDA Stream 
    for (int i = 0; i < N_CUDA_STREAMS; i++)
		CUDA_CHECK_ERROR(hipStreamCreate(&stream[i]));
}

void CudaOCTProcess::setCufftPlan()
{
	// FFT Plans 
	int rank = 1;
	int n1[] = { nScansFFT };
	int n2[] = { n2ScansFFT };
	int istride = 1, ostride = 1;
	int idist1 = nScansFFT, odist1 = idist1;
	int idist2 = n2ScansFFT, odist2 = idist2;
	int inembed[] = { 0 }, onembed[] = { 0 };
    int batch = nAlines / N_CUDA_PARTITIONS / N_CUDA_STREAMS;
	
    for (int i = 0; i < N_CUDA_STREAMS; i++)
	{
		CUFFT_CHECK_ERROR(hipfftPlanMany(&cufftPlan_R2C[i], rank, n1, inembed, istride, idist1, onembed, ostride, odist1, HIPFFT_R2C, batch));
		CUFFT_CHECK_ERROR(hipfftSetStream(cufftPlan_R2C[i], stream[i]));

		CUFFT_CHECK_ERROR(hipfftPlanMany(&cufftPlan_C2C_Inv[i], rank, n1, inembed, istride, idist1, onembed, ostride, odist1, HIPFFT_C2C, batch));
		CUFFT_CHECK_ERROR(hipfftSetStream(cufftPlan_C2C_Inv[i], stream[i]));

		CUFFT_CHECK_ERROR(hipfftPlanMany(&cufftPlan_C2C_Fwd[i], rank, n2, inembed, istride, idist2, onembed, ostride, odist2, HIPFFT_C2C, batch));
		CUFFT_CHECK_ERROR(hipfftSetStream(cufftPlan_C2C_Fwd[i], stream[i]));
	}
}


void CudaOCTProcess::freeMemory()
{
	// Free Objects and Memories
    for (int i = 0; i < N_CUDA_STREAMS; i++)
	{
		CUDA_CHECK_ERROR(hipStreamDestroy(stream[i]));
		CUFFT_CHECK_ERROR(hipfftDestroy(cufftPlan_R2C[i]));
		CUFFT_CHECK_ERROR(hipfftDestroy(cufftPlan_C2C_Inv[i]));
		CUFFT_CHECK_ERROR(hipfftDestroy(cufftPlan_C2C_Fwd[i]));
	}

	CUDA_CHECK_ERROR(hipFree(deviceRawFringeU16));
	CUDA_CHECK_ERROR(hipFree(deviceRawFringeF32));

	CUDA_CHECK_ERROR(hipFree(deviceBScan));
	CUDA_CHECK_ERROR(hipFree(deviceDemodulBScan));

	CUDA_CHECK_ERROR(hipFree(deviceDemodulSignal));
	CUDA_CHECK_ERROR(hipFree(deviceCalibSignal));

	CUDA_CHECK_ERROR(hipFree(deviceCalibBScan));
	CUDA_CHECK_ERROR(hipFree(deviceScaledBScan));
}

#endif
